#include "linalgops.hpp"

namespace axiom {

template <>
void gpu_dot<float>(const int n, const float* x, const float* y,
    float* out) {
  CUBLAS_CHECK(hipblasSdot(Axiom::cublas_handle(), n, x, 1, y, 1, out));
}
template <>
void gpu_dot<double>(const int n, const double* x, const double* y,
    double * out) {
  CUBLAS_CHECK(hipblasDdot(Axiom::cublas_handle(), n, x, 1, y, 1, out));
}

template <>
void gpu_asum<float>(const int n, const float* x, float* y) {
  CUBLAS_CHECK(hipblasSasum(Axiom::cublas_handle(), n, x, 1, y));
}
template <>
void gpu_asum<double>(const int n, const double* x, double* y) {
  CUBLAS_CHECK(hipblasDasum(Axiom::cublas_handle(), n, x, 1, y));
}

} /* namespace axiom */
